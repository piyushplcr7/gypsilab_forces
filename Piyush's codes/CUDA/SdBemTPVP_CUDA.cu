#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include <cassert>
#include <iostream>
#include <memory>
#include <eigen3/Eigen/Dense>

__device__ double atomicAdd(double *address, double val)
{
    unsigned long long int *address_as_ull =
        (unsigned long long int *)address;
    unsigned long long int old = *address_as_ull, assumed;

    do
    {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                                             __longlong_as_double(assumed)));

        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

__device__ double SLKernel(Eigen::Vector3d X, Eigen::Vector3d Y, Eigen::Vector3d YmX)
{
    // return 1;
    return 1. / (4 * M_PI) / YmX.norm();
}

__device__ Eigen::Vector3d Vel(Eigen::Vector3d X)
{
    // return Eigen::Vector3d(1, 0, 0);
    return Eigen::Vector3d(X(0) * X(1) * X(2), 0, 0);
}

__device__ Eigen::Matrix3d DVel(Eigen::Vector3d X)
{
    Eigen::Matrix3d out;
    out << X(1) * X(2), X(0) * X(2), X(0) * X(1), 0, 0, 0, 0, 0, 0;
    return out;
    // return Eigen::Matrix3d::Zero(3, 3);
}

__device__ double KernelA1(Eigen::Vector3d X, Eigen::Vector3d Y, Eigen::Vector3d YmX)
{
    double znorm = YmX.norm();
    return YmX.dot(Vel(X) - Vel(Y)) / (4 * M_PI) / (znorm * znorm * znorm);
}

__device__ double KernelA2(Eigen::Vector3d X, Eigen::Vector3d Y, Eigen::Vector3d YmX)
{
    return 1. / (4 * M_PI) / YmX.norm();
}

__device__ Eigen::Vector3d KernelC1(Eigen::Vector3d X, Eigen::Vector3d Y, Eigen::Vector3d YmX)
{
    // return 1;
    double znorm = YmX.norm();
    return YmX / (4 * M_PI) / (znorm * znorm * znorm);
}

__device__ Eigen::Vector3d KernelC3(Eigen::Vector3d X, Eigen::Vector3d Y, Eigen::Vector3d YmX)
{
    // return 1;
    double znorm = YmX.norm();
    return -3. / (4 * M_PI) * YmX * YmX.dot(Vel(Y) - Vel(X)) / (znorm * znorm * znorm * znorm * znorm) + 1. / (4 * M_PI) * (Vel(Y) - Vel(X)) / (znorm * znorm * znorm);
}

__device__ Eigen::Vector3d DLKernel(Eigen::Vector3d X, Eigen::Vector3d Y, Eigen::Vector3d YmX)
{
    // return 1;
    double znorm = YmX.norm();
    return -YmX / (4 * M_PI) / (znorm * znorm * znorm);
}

// Returns Intersection, DiffI, DiffJ
__device__ void IntersectionDiff(int *EltI, int *EltJ, int intersection[], int diffI[], int diffJ[])
{
    bool EltITracker[] = {false, false, false};
    bool EltJTracker[] = {false, false, false};

    for (int i = 0; i < 3; ++i)
    {
        for (int j = 0; j < 3; ++j)
        {
            if (EltI[i] == EltJ[j])
            {
                EltITracker[i] = true;
                EltJTracker[j] = true;
            }
        }
    }

    int interidx = 0, diffiidx = 0, diffjidx = 0;

    for (int i = 0; i < 3; ++i)
    {
        if (EltITracker[i])
            intersection[interidx++] = EltI[i];
        else
            diffI[diffiidx++] = EltI[i];

        if (!EltJTracker[i])
            diffJ[diffjidx++] = EltJ[i];
    }

    /* for (int i = 0; i < 3; ++i)
    {
        if (EltJTracker[i])
            intersection[interidx++] = EltJ[i];
        else
            diffJ[diffjidx++] = EltJ[i];

        if (!EltITracker[i])
            diffI[diffiidx++] = EltI[i];
    } */
}

__global__ void computeShapeDerivative(int TrialDim, int TestDim, int NTriangles, int NVertices, int NInteractions,
                                       int NThreads, const int *I, const int *J, const int *relation,
                                       const double *W0, const double *X0, int Nq0,
                                       const double *W1, const double *X1, int Nq1,
                                       const double *W2, const double *X2, int Nq2,
                                       const double *W3, const double *X3, int Nq3,
                                       double *A1, double *A2, double *C1, double *C2, double *C3, double *N,
                                       const double *TdA, const double *TnA,
                                       const int *Elements, const double *Vertices, const double *Normals, const double *Areas,
                                       const int *Elt2DofTest, const int *Elt2DofTrial,
                                       int TrialSpace, int TestSpace, int TrialOperator, int TestOperator,
                                       int NRSFTrial, int NRSFTest)
/* double *testout, double *testABCi, double *testABCj,
int *orig_elti, int *orig_eltj, int *modif_elti, int *modif_eltj) */
{
    int ThreadID = blockIdx.x * blockDim.x + threadIdx.x;

    //*shapeDerivative = 3.145;

    // Size of the matrix
    // int NInteractions = TrialDim * TestDim;

    // Number of element interaction
    // int NInteractions = NTriangles * NTriangles;

    // Matrix size is NTriangles^2. Each entry is assigned to a thread
    // InteractionsPerThread gives the max no.of element interactions assigned
    // to a thread
    int InteractionsPerThread = ceil(double(NInteractions) / double(NThreads));

    if (blockIdx.x == 0 && threadIdx.x == 0)
    {
        printf("Number of blocks: %d \n ", gridDim.x);
        printf("Threads per block: %d \n ", blockDim.x);
        printf("Total interactions: %d , Interactions per thread: %d \n ", NInteractions, InteractionsPerThread);
    }

    // Looping over all assigned interactions
    for (int idx = 0; idx < InteractionsPerThread; ++idx)
    {
        if (blockIdx.x == 0 && threadIdx.x == 0)
        {
            printf("In block 0 thread 0 computing interaction no. : %d \n ", idx);
        }
        // The interaction number
        // int InteractionIdx = ThreadID + NThreads * idx;
        int InteractionIdx = ThreadID * InteractionsPerThread + idx;

        // Preparing variables
        Eigen::Vector3d Ai, Bi, Ci, Aj, Bj, Cj;
        Eigen::MatrixXd Ei(3, 2), Ej(3, 2);

        int intersection[3], diffI[3], diffJ[3];

        const double *W = NULL;
        const double *X = NULL;
        int NQudPts = 0;

        // Make sure that the last thread stays in limit
        if (InteractionIdx >= NInteractions)
            break;

        // The pair of panels
        int i = I[InteractionIdx], j = J[InteractionIdx];

        /* printf("Interaction  (%d, %d) \n", i, j); */

        double g_tau = 2 * Areas[i], g_t = 2 * Areas[j];

        // Obtaining the normals
        /* Eigen::Vector3d normalx(Normals[i], Normals[i + NTriangles], Normals[i + 2 * NTriangles]);
        Eigen::Vector3d normaly(Normals[j], Normals[j + NTriangles], Normals[j + 2 * NTriangles]); */

        Eigen::Vector3d normalx(Normals[3 * i], Normals[3 * i + 1], Normals[3 * i + 2]);
        Eigen::Vector3d normaly(Normals[3 * j], Normals[3 * j + 1], Normals[3 * j + 2]);

        /* int EltI[] = {Elements[i],
                      Elements[i + NTriangles],
                      Elements[i + 2 * NTriangles]};

        int EltJ[] = {Elements[j],
                      Elements[j + NTriangles],
                      Elements[j + 2 * NTriangles]}; */

        int EltI[] = {Elements[3 * i],
                      Elements[3 * i + 1],
                      Elements[3 * i + 2]};

        int EltJ[] = {Elements[3 * j],
                      Elements[3 * j + 1],
                      Elements[3 * j + 2]};

        int origEltI[] = {EltI[0], EltI[1], EltI[2]};
        int origEltJ[] = {EltJ[0], EltJ[1], EltJ[2]};

        /* int DofsI[] = {Elt2DofTest[i], Elt2DofTest[i + NTriangles], Elt2DofTest[i + 2 * NTriangles]};
        int DofsJ[] = {Elt2DofTrial[j], Elt2DofTrial[j + NTriangles], Elt2DofTrial[j + 2 * NTriangles]}; */

        int DofsI[] = {Elt2DofTest[3 * i], Elt2DofTest[3 * i + 1], Elt2DofTest[3 * i + 2]};
        int DofsJ[] = {Elt2DofTrial[3 * j], Elt2DofTrial[3 * j + 1], Elt2DofTrial[3 * j + 2]};

        // Original permutation of elements
        int permI[] = {0, 1, 2};
        int permJ[] = {0, 1, 2};

        if (relation[InteractionIdx] == 0) // No interaction
        {
            // Computing Quadrature
            W = W0;
            X = X0;
            NQudPts = Nq0;
        }
        else if (relation[InteractionIdx] == 1) // Common vertex
        {
            IntersectionDiff(EltI, EltJ, intersection, diffI, diffJ);

            for (int l = 0; l < 3; ++l)
            {
                // Permutation for I
                if (EltI[l] == intersection[0])
                {
                    permI[0] = l;
                }
                else if (EltI[l] == diffI[0])
                {
                    permI[1] = l;
                }
                else if (EltI[l] == diffI[1])
                {
                    permI[2] = l;
                }

                // Permutation for J
                if (EltJ[l] == intersection[0])
                {
                    permJ[0] = l;
                }
                else if (EltJ[l] == diffJ[0])
                {
                    permJ[1] = l;
                }
                else if (EltJ[l] == diffJ[1])
                {
                    permJ[2] = l;
                }
            }

            // Changing EltI into ABCI
            EltI[0] = intersection[0];
            EltI[1] = diffI[0];
            EltI[2] = diffI[1];

            // Changing EltI into ABCJ
            EltJ[0] = intersection[0];
            EltJ[1] = diffJ[0];
            EltJ[2] = diffJ[1];

            // Computing Quadrature
            W = W1;
            X = X1;
            NQudPts = Nq1;
        }
        else if (relation[InteractionIdx] == 2) // Common edge
        {
            IntersectionDiff(EltI, EltJ, intersection, diffI, diffJ);

            for (int l = 0; l < 3; ++l)
            {
                // Permutation for I
                if (EltI[l] == intersection[0])
                {
                    permI[0] = l;
                }
                else if (EltI[l] == intersection[1])
                {
                    permI[1] = l;
                }
                else if (EltI[l] == diffI[0])
                {
                    permI[2] = l;
                }

                // Permutation for J
                if (EltJ[l] == intersection[0])
                {
                    permJ[0] = l;
                }
                else if (EltJ[l] == intersection[1])
                {
                    permJ[1] = l;
                }
                else if (EltJ[l] == diffJ[0])
                {
                    permJ[2] = l;
                }
            }

            EltI[0] = intersection[0];
            EltI[1] = intersection[1];
            EltI[2] = diffI[0];

            EltJ[0] = intersection[0];
            EltJ[1] = intersection[1];
            EltJ[2] = diffJ[0];

            // Computing Quadrature
            W = W2;
            X = X2;
            NQudPts = Nq2;
        }
        else // Identical panels, case 3
        {
            // Computing Quadrature
            W = W3;
            X = X3;
            NQudPts = Nq3;
        }

        // Vertices of element i
        Ai = Eigen::Vector3d(Vertices[3 * EltI[0]], Vertices[3 * EltI[0] + 1], Vertices[3 * EltI[0] + 2]);
        Bi = Eigen::Vector3d(Vertices[3 * EltI[1]], Vertices[3 * EltI[1] + 1], Vertices[3 * EltI[1] + 2]);
        Ci = Eigen::Vector3d(Vertices[3 * EltI[2]], Vertices[3 * EltI[2] + 1], Vertices[3 * EltI[2] + 2]);

        // Vertices of element j
        Aj = Eigen::Vector3d(Vertices[3 * EltJ[0]], Vertices[3 * EltJ[0] + 1], Vertices[3 * EltJ[0] + 2]);
        Bj = Eigen::Vector3d(Vertices[3 * EltJ[1]], Vertices[3 * EltJ[1] + 1], Vertices[3 * EltJ[1] + 2]);
        Cj = Eigen::Vector3d(Vertices[3 * EltJ[2]], Vertices[3 * EltJ[2] + 1], Vertices[3 * EltJ[2] + 2]);

        // Jacobian Matrices

        Ei.col(0) = Bi - Ai;
        Ei.col(1) = Ci - Ai;

        Ej.col(0) = Bj - Aj;
        Ej.col(1) = Cj - Aj;

        Eigen::MatrixXd EtEi = Ei.transpose() * Ei;
        Eigen::MatrixXd EtEj = Ej.transpose() * Ej;
        double deti = EtEi(0, 0) * EtEi(1, 1) - EtEi(0, 1) * EtEi(1, 0);
        double detj = EtEj(0, 0) * EtEj(1, 1) - EtEj(0, 1) * EtEj(1, 0);

        Eigen::MatrixXd Dxyi = -EtEi;
        Eigen::MatrixXd Dxyj = -EtEj;

        Dxyi(0, 0) = EtEi(1, 1);
        Dxyi(1, 1) = EtEi(0, 0);

        Dxyj(0, 0) = EtEj(1, 1);
        Dxyj(1, 1) = EtEj(0, 0);

        Dxyi /= deti;
        Dxyj /= detj;

        Eigen::MatrixXd DCVi = Ei * Dxyi, DCVj = Ej * Dxyj;

        Eigen::Matrix3d LocalMatrix = Eigen::MatrixX3d::Zero(3, 3);
        Eigen::Matrix3d LocalMatrixA1 = Eigen::MatrixX3d::Zero(3, 3);
        Eigen::Matrix3d LocalMatrixA2 = Eigen::MatrixX3d::Zero(3, 3);
        Eigen::Matrix3d LocalMatrixC1 = Eigen::MatrixX3d::Zero(3, 3);
        Eigen::Matrix3d LocalMatrixC3 = Eigen::MatrixX3d::Zero(3, 3);
        Eigen::Matrix3d LocalMatrixN = Eigen::MatrixX3d::Zero(3, 3);

        // P0 X P0
        /* for (int ii = 0; ii < NRSFTest; ++ii)
        {
            double Psix = g_tau;
            for (int jj = 0; jj < NRSFTrial; ++jj)
            {
                double Psiy = g_t;
                for (int QudPt = 0; QudPt < NQudPts; ++QudPt)
                {
                    Eigen::Vector3d chi_tau = Ai + Ei.col(0) * X[QudPt] + Ei.col(1) * X[QudPt + NQudPts];
                    Eigen::Vector3d chi_t = Aj + Ej.col(0) * X[QudPt + 2 * NQudPts] + Ej.col(1) * X[QudPt + 3 * NQudPts];
                    LocalMatrix(ii, jj) += W[QudPt] * Psix * SLKernel(chi_tau, chi_t, chi_t - chi_tau) * Psiy;
                }
                GalerkinMatrix[i + TestDim * j] += LocalMatrix(ii, jj);

                // Atomic update of the galerkin matrix
                // double contribution = LocalMatrix(ii, jj);
                // atomicAdd(&GalerkinMatrix[i + TestDim * j], contribution);
            }
        } */

        // P1 X P1
        /* for (int ii = 0; ii < NRSFTest; ++ii)
        {

            for (int jj = 0; jj < NRSFTrial; ++jj)
            {

                for (int QudPt = 0; QudPt < NQudPts; ++QudPt)
                {
                    Eigen::Vector3d RSFsX(1 - X[4 * QudPt] - X[4 * QudPt + 1], X[4 * QudPt], X[4 * QudPt + 1]);
                    Eigen::Vector3d RSFsY(1 - X[4 * QudPt + 2] - X[4 * QudPt + 3], X[4 * QudPt + 2], X[4 * QudPt + 3]);

                    RSFsX *= g_tau;
                    RSFsY *= g_t;

                    Eigen::Vector3d chi_tau = Ai + Ei.col(0) * X[4 * QudPt] + Ei.col(1) * X[4 * QudPt + 1];
                    Eigen::Vector3d chi_t = Aj + Ej.col(0) * X[4 * QudPt + 2] + Ej.col(1) * X[4 * QudPt + 3];

                    LocalMatrix(ii, jj) += W[QudPt] * RSFsX(ii) * SLKernel(chi_tau, chi_t, chi_t - chi_tau) * RSFsY(jj);
                }
                // GalerkinMatrix[EltI[ii] + TestDim * EltJ[jj]] += LocalMatrix(ii, jj);

                //  Atomic update of the galerkin matrix
                atomicAdd(&GalerkinMatrix[EltI[ii] + TestDim * EltJ[jj]], LocalMatrix(ii, jj));
            }
        } */

        // P0 X ntimes(P1)
        /* for (int ii = 0; ii < NRSFTest; ++ii)
        {
            double Psix = g_tau;

            for (int jj = 0; jj < NRSFTrial; ++jj)
            {

                for (int QudPt = 0; QudPt < NQudPts; ++QudPt)
                {

                    Eigen::Vector3d RSFsY(1 - X[QudPt + 2 * NQudPts] - X[QudPt + 3 * NQudPts], X[QudPt + 2 * NQudPts], X[QudPt + 3 * NQudPts]);

                    RSFsY *= g_t;

                    Eigen::Vector3d chi_tau = Ai + Ei.col(0) * X[QudPt] + Ei.col(1) * X[QudPt + NQudPts];
                    Eigen::Vector3d chi_t = Aj + Ej.col(0) * X[QudPt + 2 * NQudPts] + Ej.col(1) * X[QudPt + 3 * NQudPts];

                    LocalMatrix(ii, jj) += W[QudPt] * Psix * DLKernel(chi_tau, chi_t, chi_t - chi_tau).dot(normaly) * RSFsY(jj);
                }
                // GalerkinMatrix[i + TestDim * EltJ[jj]] += LocalMatrix(ii, jj);

                //  Atomic update of the galerkin matrix
                atomicAdd(&GalerkinMatrix[i + TestDim * EltJ[jj]], LocalMatrix(ii, jj));
            }
        } */

        // RWG X RWG
        for (int ii = 0; ii < 3; ++ii)
        {
            int iip1 = (permI[ii] + 1) % 3;
            int iip2 = (iip1 + 1) % 3;

            double fluxI = origEltI[iip1] < origEltI[iip2] ? 1. : -1.;
            // double RWGX_ref_0 = -ii % 2, RWGX_ref_1 = -ii / 2;

            for (int jj = 0; jj < 3; ++jj)
            {
                int jjp1 = (permJ[jj] + 1) % 3;
                int jjp2 = (jjp1 + 1) % 3;

                double fluxJ = origEltJ[jjp1] < origEltJ[jjp2] ? 1. : -1.;
                // double RWGY_ref_0 = -jj % 2, RWGY_ref_1 = -jj / 2;

                for (int QudPt = 0; QudPt < NQudPts; ++QudPt)
                {
                    /* if (blockIdx.x == 0 && threadIdx.x == 0)
                    {
                        printf("Qud pt %d\n", QudPt);
                    } */
                    // Reference basis RT0
                    /* Eigen::MatrixXd RWGX_ref(3, 2); // Rows represent the 3 RSFs
                    RWGX_ref << X[4 * QudPt], X[4 * QudPt + 1],
                        X[4 * QudPt] - 1, X[4 * QudPt + 1],
                        X[4 * QudPt], X[4 * QudPt + 1] - 1;

                    Eigen::MatrixXd RWGY_ref(3, 2); // Rows represent the 3 RSFs
                    RWGY_ref << X[4 * QudPt + 2], X[4 * QudPt + 3],
                        X[4 * QudPt + 2] - 1, X[4 * QudPt + 3],
                        X[4 * QudPt + 2], X[4 * QudPt + 3] - 1;

                    Eigen::Vector3d Psix = fluxI * Ei * RWGX_ref.row(ii).transpose();
                    Eigen::Vector3d Psiy = fluxJ * Ej * RWGY_ref.row(jj).transpose(); */

                    double RWGX_ref_0 = X[4 * QudPt] - ii % 2;
                    double RWGX_ref_1 = X[4 * QudPt + 1] - ii / 2;

                    double RWGY_ref_0 = X[4 * QudPt + 2] - jj % 2;
                    double RWGY_ref_1 = X[4 * QudPt + 3] - jj / 2;

                    // RWG elements
                    Eigen::Vector3d Psix = fluxI * (Ei.col(0) * RWGX_ref_0 + Ei.col(1) * RWGX_ref_1);
                    Eigen::Vector3d Psiy = fluxJ * (Ej.col(0) * RWGY_ref_0 + Ej.col(1) * RWGY_ref_1);

                    Eigen::Vector3d chi_tau = Ai + Ei.col(0) * X[4 * QudPt] + Ei.col(1) * X[4 * QudPt + 1];
                    Eigen::Vector3d chi_t = Aj + Ej.col(0) * X[4 * QudPt + 2] + Ej.col(1) * X[4 * QudPt + 3];

                    // A1 with RWG . RWG
                    LocalMatrixA1(ii, jj) += W[QudPt] * KernelA1(chi_tau, chi_t, chi_t - chi_tau) * Psiy.dot(Psix);

                    // A2 with DVelRWG(y) . RWG(x)
                    LocalMatrixA2(ii, jj) += W[QudPt] * KernelA2(chi_tau, chi_t, chi_t - chi_tau) * (DVel(chi_t) * Psiy).dot(Psix);

                    // C1 with DVelRWG(y) X RWG(X)
                    LocalMatrixC1(ii, jj) += W[QudPt] * (KernelC1(chi_tau, chi_t, chi_t - chi_tau).cross(DVel(chi_t) * Psiy)).dot(Psix);

                    // C3 with RWG X RWG
                    LocalMatrixC3(ii, jj) += W[QudPt] * (KernelC3(chi_tau, chi_t, chi_t - chi_tau).cross(Psiy)).dot(Psix);

                    // N with RWG.div X RWG.div, kernelN = kernelA1
                    LocalMatrixN(ii, jj) += W[QudPt] * KernelA1(chi_tau, chi_t, chi_t - chi_tau) * 4 * fluxI * fluxJ;
                }
                // GalerkinMatrix[i + TestDim * EltJ[jj]] += LocalMatrix(ii, jj);

                // Accumulating the shape derivative
                // Local matrix (ii,jj) contains part of the global element DofsI[permI[ii]] , DofsJ[permJ[jj]]

                // TnA' * A1mat * TnA
                atomicAdd(A1, TnA[DofsI[permI[ii]]] * LocalMatrixA1(ii, jj) * TnA[DofsJ[permJ[jj]]]);

                // 2 * TnA' * A2mat * TnA
                atomicAdd(A2, 2 * TnA[DofsI[permI[ii]]] * LocalMatrixA2(ii, jj) * TnA[DofsJ[permJ[jj]]]);

                // C1 = TnA' * C1mat * TdA
                atomicAdd(C1, TnA[DofsI[permI[ii]]] * LocalMatrixC1(ii, jj) * TdA[DofsJ[permJ[jj]]]);

                // C2 = TdA' * C1mat * TnA
                atomicAdd(C2, TdA[DofsI[permI[ii]]] * LocalMatrixC1(ii, jj) * TnA[DofsJ[permJ[jj]]]);

                // C3 = TnA' * C3mat * TdA
                atomicAdd(C3, TnA[DofsI[permI[ii]]] * LocalMatrixC3(ii, jj) * TdA[DofsJ[permJ[jj]]]);

                // N = -TdA' * Nmat * TdA
                atomicAdd(N, -TdA[DofsI[permI[ii]]] * LocalMatrixN(ii, jj) * TdA[DofsJ[permJ[jj]]]);

                //  Atomic update of the galerkin matrix
                // atomicAdd(&GalerkinMatrix[DofsI[permI[ii]] + TestDim * DofsJ[permJ[jj]]], LocalMatrix(ii, jj));
            }
        }
    }
}