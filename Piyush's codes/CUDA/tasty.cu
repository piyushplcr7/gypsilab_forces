#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include <cassert>
#include <iostream>
#include <memory>
#include <eigen3/Eigen/Dense>

__global__ void add1(double *a, double b)
{
    *a += b;
}